#include "utility.cuh"

#include <string>
#include <stdexcept>

void HandleError(hipError_t err, const char *string, const char *file, int line) {
  if (err != hipSuccess) {
    //printf("%s\n", string);
    //printf("%s in \n\n%s at line %d\n", hipGetErrorString(err), file, line);
    throw std::runtime_error(
        std::string("CUDA Error ") + hipGetErrorString(err) + " " + string + " in " + file + " at line "
            + std::to_string(line));
  }
}
void HandleError(const char *file, int line) {
  auto err = hipGetLastError();
  if (err != hipSuccess) {
    //printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
    throw std::runtime_error(
        std::string("CUDA Error ") + hipGetErrorString(err) + " in " + file + " at line " + std::to_string(line));
  }
}
float end_clock(hipEvent_t &start, hipEvent_t &end) {
  float time;
  HANDLE_ERROR(hipEventRecord(end, 0));
  HANDLE_ERROR(hipEventSynchronize(end));
  HANDLE_ERROR(hipEventElapsedTime(&time, start, end));

  // Returns ms
  return time;
}

void start_clock(hipEvent_t &start) {
  HANDLE_ERROR(hipEventRecord(start, 0));
}
